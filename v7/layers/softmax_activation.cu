#include "hip/hip_runtime.h"
#include "softmax_activation.hh"
#include "../nn_utils/nn_exception.hh"
#include <iostream>

__inline__ __device__ float expsum(float* Z, int Z_y_dim, int row) {
    float psum = 0;
    for (int i = 0; i < Z_y_dim; i++) {
        psum += expf(Z[row * Z_y_dim + i]);
    }
    return psum;
}

__inline__ __device__ float softmax(float Z_i, float esum) {
    return expf(Z_i) / esum;
}

__global__ void softmaxActivationForward(float* Z, float* A, 
                                         int Z_x_dim, int Z_y_dim) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    if (n < Z_x_dim * Z_y_dim) {
        int r = n / Z_y_dim;
        A[n] = softmax(Z[n], expsum(Z, Z_y_dim, r));
    }
}

__global__ void softmaxActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    if (n < Z_x_dim * Z_y_dim) {
        int r = n / Z_y_dim;
        float smax = softmax(Z[n], expsum(Z, Z_y_dim, r));
        dZ[n] = dA[n] * smax * (1.f - smax);
    }
}

SoftmaxActivation::SoftmaxActivation(std::string name) {
    this->name = name;
}

SoftmaxActivation::~SoftmaxActivation() {}

Matrix& SoftmaxActivation::forward(Matrix& Z) {
    this->Z = Z;
    A.allocateMemoryIfNotAllocated(Z.shape);

    dim3 G(256);
    dim3 B((Z.shape.y * Z.shape.x + G.x - 1) / G.x);

    softmaxActivationForward <<< B, G >>> (Z.data_device.get(), A.data_device.get(), Z.shape.x, Z.shape.y);

    NNException::throwIfDeviceErrorsOccurred("Cannot perform softmax forward propagation.");

    return A;
}

Matrix& SoftmaxActivation::backprop(Matrix& dA, float learning_rate) {
    dZ.allocateMemoryIfNotAllocated(Z.shape);

    dim3 G(256);
    dim3 B((Z.shape.y * Z.shape.x + G.x - 1) / G.x);

    softmaxActivationBackprop <<< B, G >>> (Z.data_device.get(), dA.data_device.get(), dZ.data_device.get(), Z.shape.x, Z.shape.y);

    NNException::throwIfDeviceErrorsOccurred("Cannot perform softmax back propagation");

    return dZ;
}
