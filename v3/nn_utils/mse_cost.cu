#include "hip/hip_runtime.h"
#include "mse_cost.hh"
#include "nn_exception.hh"

#include <assert.h>
#include <iostream>
#include <math.h>

__global__ void meanSquareErrorCost(float* predictions, float* target, int N, int C, float* cost) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    if (n < N) {
        float sum = 0.0f;
        // TODO: more intelligent reduction
        for (int c = 0; c < C; c++) {
            int idx = n * C + c;
            float diff = predictions[idx] - (c == static_cast<int>(target[n]));
            sum += diff * diff;
        }
        atomicAdd(cost, sum / C);
    }
}

__global__ void dMeanSquareErrorCost(float* predictions, float* target, float* dY, int N, int C) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    if (n < N) {
        for (int c = 0; c < C; c++) {
            int idx = n * C + c;
            dY[idx] = 2.0f * (predictions[idx] - (c == static_cast<int>(target[n]))) / C;
        }
    }
}

MSECost::MSECost() {}

MSECost::~MSECost() {}

float MSECost::cost(Matrix predictions, Matrix target) {
    assert(predictions.shape.x == target.shape.x);

    float* cost;
    float* d_cost;
    hipMalloc(&d_cost, sizeof(float));
    hipMemset(d_cost, 0, sizeof(float));

    dim3 T(64);
    dim3 B((predictions.shape.y * predictions.shape.x + T.x - 1) / T.x);
    meanSquareErrorCost<<< B, T >>>(
        predictions.data_device.get(), target.data_device.get(),
        predictions.shape.x, predictions.shape.y, d_cost);
    hipDeviceSynchronize();
    NNException::throwIfDeviceErrorsOccurred(
        "Cannot compute MSE cost.");

    hipMemcpy(cost, d_cost, 1*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_cost);

    return *cost;
}

Matrix MSECost::dCost(Matrix predictions, Matrix target, Matrix dY) {
    assert(predictions.shape.x == target.shape.x);

    dim3 block_size(256);
    // dim3 block_size(32, 32);
    dim3 num_of_blocks((predictions.shape.y * predictions.shape.x + block_size.x - 1) / block_size.x);
    dMeanSquareErrorCost<<<num_of_blocks, block_size>>>(
        predictions.data_device.get(), target.data_device.get(),
        dY.data_device.get(), predictions.shape.x, predictions.shape.y);
    NNException::throwIfDeviceErrorsOccurred(
        "Cannot compute derivative for mean square error.");

    return dY;
}
